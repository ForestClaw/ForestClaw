#include "hip/hip_runtime.h"
#include "update_q.h"
__global__ void update_q_cuda(int x_stride, int mbc, double dtdx, double dtdy,
                              double* qold, double* fm, double* fp, double* gm,
                              double* gp) {
    int i = (blockIdx.x + mbc) * x_stride + blockIdx.y + mbc;
    qold[i] += -dtdx * (fm[i+x_stride] - fp[i]) -
                          dtdy * (gm[i+1] - gp[i]);
}
void update_q_(int& meqn, int& mx, int& my, int& mbc, double& dtdx,
               double& dtdy, double* qold, double* fm, double* fp, double* gm,
               double* gp, int& mcapa) {
    int size = meqn * (mx + 2 * mbc) * (my + 2 * mbc);
    double* qold_dev;
    double* fm_dev;
    double* fp_dev;
    double* gm_dev;
    double* gp_dev;
    hipMalloc((void**)&qold_dev, size * sizeof(double));
    hipMalloc((void**)&fm_dev, size * sizeof(double));
    hipMalloc((void**)&fp_dev, size * sizeof(double));
    hipMalloc((void**)&gm_dev, size * sizeof(double));
    hipMalloc((void**)&gp_dev, size * sizeof(double));
    hipMemcpy(qold_dev, qold, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fm_dev, fm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(fp_dev, fp, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gm_dev, gm, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gp_dev, gp, size * sizeof(double), hipMemcpyHostToDevice);

    // this is not optimal
    dim3 dimGrid(mx, my);
    dim3 dimBlock(1, 1);

    int x_stride = mx + 2 * mbc;
    update_q_cuda<<<dimGrid, dimBlock>>>(x_stride, mbc, dtdx, dtdy, qold_dev,
                                         fm_dev, fp_dev, gm_dev, gp_dev);

    hipMemcpy(qold, qold_dev, size * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(qold_dev);
    hipFree(fm_dev);
    hipFree(fp_dev);
    hipFree(gm_dev);
    hipFree(gp_dev);
}
