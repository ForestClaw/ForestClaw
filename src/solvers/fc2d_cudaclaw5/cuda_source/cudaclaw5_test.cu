#include "hip/hip_runtime.h"
#include "../fc2d_cudaclaw5.h"

__global__ void donothing()
{
    /* Do nothing! */

    return;
}

__device__ int addem( int a, int b ) 
{
    return a + b;
}

__global__ void add( int a, int b, int *c ) 
{
    *c = addem( a, b );
}


void cudaclaw5_test()
{
    int a,b,c;
    int *dev_c;

    donothing<<<1,1>>>();

    /* Allocate memory on the device */
    hipMalloc( (void**)&dev_c, sizeof(int));

    a = 2;
    b = 7;
    add<<<1,1>>>(a, b, dev_c );   

    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
 
    printf("Kernel result : %d + %d = %d\n",a,b,c);   

    hipFree( dev_c);

    return;
}